
#include <hip/hip_runtime.h>
__global__ void add_vectors(
    const int* a,
    const int* b,
    int *c,
    const int n)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= n) return;

    c[idx] = a[idx] + b[idx];
}
